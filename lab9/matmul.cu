#include "hip/hip_runtime.h"
#include "matmul.h"

int main(int argc, char * const argv[])
{
  int size;
  Matrix A, B, C_gpu, C_gpu_blas, C_cpu;
  float gpu_time, gpu_blas_time, cpu_time;

  if (argc != 2) {
    printf("usage: matmul SIZE\n");
    exit(1);
  }

  size = atoi(argv[1]);

  A.width = size;
  A.height = size;
  A.elements = new float[size * size];

  B.width = size;
  B.height = size;
  B.elements = new float[size * size];

  C_gpu.width = size;
  C_gpu.height = size;
  C_gpu.elements = new float[size * size];

  C_gpu_blas.width = size;
  C_gpu_blas.height = size;
  C_gpu_blas.elements = new float[size * size];

  C_cpu.width = size;
  C_cpu.height = size;
  C_cpu.elements = new float[size * size];

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      A.elements[i * size + j] = rand() % 100;
      B.elements[i * size + j] = rand() % 100;
    }
  }

  gpu_time = matmul_gpu(A, B, C_gpu);
  gpu_blas_time = matmul_gpu_cublas(A, B, C_gpu_blas);
  cpu_time = matmul_cpu(A, B, C_cpu);

  // printf("GPU time: %f\n", gpu_time);
  // printf("GPU CUBLAS time: %f\n", gpu_blas_time);
  // printf("CPU time: %f\n", cpu_time);
  //
  // printf("Check GPU: %d\n", check(C_gpu, C_cpu));
  // printf("Check GPU CUBLAS: %d\n", check(C_gpu_blas, C_cpu));

  printf("%d,%f,%f,%f\n", size, gpu_time, gpu_blas_time, cpu_time);
}

// matrix GPU multiplication
float matmul_gpu(const Matrix A, const Matrix B, Matrix C)
{
  Matrix d_A, d_B, d_C;
  size_t size;
  float time;

  d_A = cudaMatrixCopy(A);
  d_B = cudaMatrixCopy(B);
  d_C = cudaMatrixCopy(C);

  // set block size
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // set grid size
  dim3 dimGrid(
    (C.width + dimBlock.x - 1) / dimBlock.x,
    (C.height + dimBlock.y - 1) / dimBlock.y
  );

  // start timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  // invoke kernel
  matmul_gpu_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // synchronize threads
  hipDeviceSynchronize();

  // stop timer, calculate total time
  sdkStopTimer(&timer);
  time = (float) sdkGetTimerValue(&timer) / 1000;
  sdkDeleteTimer(&timer);

  // copy results from device to host
  size = C.width * C.height * sizeof(float);
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // clean up device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  return time;
}

// matrix GPU multiplication kernel
__global__ void matmul_gpu_kernel(Matrix A, Matrix B, Matrix C)
{
  // each thread computes one element of product C
  // results are being accumulated in C_value

  float C_value = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // terminate thread if out of matrix bounds
  if (row >= A.height || col >= B.width) return;

  for (int i = 0; i < A.width; i++) {
    C_value += A.elements[row * A.width + i] * B.elements[i * B.width + col];
  }

  C.elements[row * C.width + col] = C_value;
}

// matrix GPU CUBLAS multiplication
float matmul_gpu_cublas(const Matrix A, const Matrix B, Matrix C) {
  Matrix d_A, d_B, d_C;
  size_t size;
  float time;

  hipEvent_t start, stop;
  hipblasHandle_t handle;

  const float alpha = 1.0;
  const float beta = 0.0;

  // load input matrices into device memory
  d_A = cudaMatrixCopy(A);
  d_B = cudaMatrixCopy(B);
  d_C = cudaMatrixCopy(C);

  // set block size
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // set grid size
  dim3 dimGrid(
    (C.width + dimBlock.x - 1) / dimBlock.x,
    (C.height + dimBlock.y - 1) / dimBlock.y
  );

  // create CUBLAS handle
  hipblasCreate(&handle);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // record start event
  hipEventRecord(start, NULL);

  // perform matrix multiplication
  hipblasSgemm(
    handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    d_B.width,
    d_A.height,
    d_A.width,
    &alpha,
    d_B.elements,
    d_B.width,
    d_A.elements,
    d_A.width,
    &beta,
    d_C.elements,
    d_A.width
  );

  // record stop event
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);

  // calculate time between events
  hipEventElapsedTime(&time, start, stop);
  time /= 1000;

  // destroy handle
  hipblasDestroy(handle);

  // copy results from device to host
  size = C.width * C.height * sizeof(float);
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // clean up device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  return time;
}

// matrix CPU multiplication
float matmul_cpu(const Matrix A, const Matrix B, Matrix C) {
  int i, j, k;
  float sum;
  clock_t start_time, end_time;

  start_time = clock();

  for (i = 0; i < A.height; i++) {
    for (j = 0; j < B.width; j++) {
      sum = 0;
      for (k = 0; k < A.width; k++) {
        sum += A.elements[i * A.width + k] * B.elements[k * B.width + j];
      }

      C.elements[i * B.width + j] = sum;
    }
  }

  end_time = clock();
  return (float) (end_time - start_time) / CLOCKS_PER_SEC;
}

// verifies if product matrices calculated by cpu and GPU are equal
int check(const Matrix A, const Matrix B) {
  if (A.width != B.width || A.height != B.height) return 0;
  for (int i = 0; i < A.width; i++) {
    for (int j = 0; j < A.height; j++) {
      if (A.elements[j * A.width + i] != B.elements[j * A.width + i])
        return 0;
    }
  }
  return 1;
}

Matrix cudaMatrixCopy(const Matrix A) {
  Matrix d_A;
  size_t size = A.width * A.height * sizeof(float);

  d_A.width = A.width;
  d_A.height = A.height;

  hipMalloc((void**) &d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  return d_A;
}
