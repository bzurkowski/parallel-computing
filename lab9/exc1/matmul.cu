#include "hip/hip_runtime.h"
#include "matmul.h"

int main(int argc, char * const argv[])
{
  int size;

  Matrix A;
  Matrix B;
  Matrix C_gpu;
  Matrix C_cpu;

  double gpu_time, cpu_time;

  if (argc != 2) {
    printf("usage: matmul SIZE\n");
    exit(1);
  }

  size = atoi(argv[1]);

  A.width = size;
  B.width = size;

  C_gpu.width = size;
  C_cpu.width = size;

  A.height = size;
  B.height = size;

  C_gpu.height = size;
  C_cpu.height = size;

  A.elements = new float[size*size];
  B.elements = new float[size*size];

  C_gpu.elements = new float[size*size];
  C_cpu.elements = new float[size*size];

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      A.elements[i * size + j] = rand() % 100;
      B.elements[i * size + j] = rand() % 100;
    }
  }

  gpu_time = matmul_gpu(A, B, C_gpu);
  cpu_time = matmul_cpu(A, B, C_cpu);

  // printf("GPU time: %f\n", gpu_time);
  // printf("CPU time: %f\n", cpu_time);
  // printf("Check: %d\n", check(C_gpu, C_cpu));
  printf("%d,%f,%f\n", size, cpu_time, gpu_time);
}

// matrix gpu multiplication
double matmul_gpu(const Matrix A, const Matrix B, Matrix C)
{
  Matrix d_A, d_B, d_C;
  size_t size;
  double time;

  d_A.width = A.width;
  d_B.width = B.width;
  d_C.width = C.width;

  d_A.height = A.height;
  d_B.height = B.height;
  d_C.height = C.height;

  // load input matrices into device memory
  size = A.width * A.height * sizeof(float);
  hipMalloc((void**) &d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  size = B.width * B.height * sizeof(float);
  hipMalloc((void**) &d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // allocate device memory for matrix product
  size = C.width * C.height * sizeof(float);
  hipMalloc((void**) &d_C.elements, size);

  // set block size
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // set grid size
  dim3 dimGrid(
    (B.width + dimBlock.x - 1) / dimBlock.x,
    (B.height + dimBlock.y - 1) / dimBlock.y
  );

  // start timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  // invoke kernel
  matmul_gpu_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // synchronize threads
  hipDeviceSynchronize();

  // stop timer, calculate total time
  sdkStopTimer(&timer);
  time = (double) sdkGetTimerValue(&timer) / 1000;
  sdkDeleteTimer(&timer);

  // copy results from device to host
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // clean up device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  return time;
}

// matrix gpu multiplication kernel
__global__ void matmul_gpu_kernel(Matrix A, Matrix B, Matrix C)
{
  // each thread computes one element of product C
  // results are being accumulated in C_value

  float C_value = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // terminate thread if out of matrix bounds
  if (row >= A.height || col >= B.width) return;

  for (int i = 0; i < A.width; i++) {
    C_value += A.elements[row * A.width + i] * B.elements[i * B.width + col];
  }

  C.elements[row * C.width + col] = C_value;
}

// matrix cpu multiplication
double matmul_cpu(const Matrix A, const Matrix B, Matrix C) {
  int i, j, k;
  float sum;
  clock_t start_time, end_time;

  start_time = clock();

  for (i = 0; i < A.height; i++) {
    for (j = 0; j < B.width; j++) {
      sum = 0;
      for (k = 0; k < A.width; k++) {
        sum += A.elements[i * A.width + k] * B.elements[k * B.width + j];
      }

      C.elements[i * B.width + j] = sum;
    }
  }

  end_time = clock();
  return (double) (end_time - start_time) / CLOCKS_PER_SEC;
}

// verifies if product matrices calculated by cpu and gpu are equal
int check(const Matrix A, const Matrix B) {
  if (A.width != B.width || A.height != B.height) return 0;
  for (int i = 0; i < A.width; i++) {
    for (int j = 0; j < A.height; j++) {
      if (A.elements[j * A.width + i] != B.elements[j * A.width + i])
        return 0;
    }
  }
  return 1;
}
