#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "helper_functions.h"


__global__ void add_vectors_gpu(int *a, int *b, int *c, unsigned long long int size);

void add_vectors_cpu(int *a, int *b, int *c, unsigned long long int size);

int *alloc_vector(unsigned long long int size);

int * random_vector(unsigned long long int size);

int check(int *c1, int *c2, unsigned long long int size);

void print_vector(int *v, unsigned long long int size);


int main(int argc, char **argv) {
  unsigned long long int size;
  int *a, *b, *c_cpu, *c_gpu;
  int *da, *db, *dc;
  int num_blocks, num_threads;
  double cpu_time, gpu_time;
  clock_t cpu_begin, cpu_end;

  if (argc != 3) {
    printf("usage: add_vectors SIZE NUM_THREADS_PER_BLOCK");
    exit(1);
  }

  srand(time(NULL));

  size = atoi(argv[1]);
  num_threads = atoi(argv[2]);
  num_blocks = (size + num_threads - 1) / num_threads;

  // generate random input vectors
  a = random_vector(size);
  b = random_vector(size);

  // allocate memory for results
  c_cpu = alloc_vector(size);
  c_gpu = alloc_vector(size);

  // add vectors on host
  cpu_begin = clock();
  add_vectors_cpu(a, b, c_cpu, size);
  cpu_end = clock();

  cpu_time = (double) (cpu_end - cpu_begin) / CLOCKS_PER_SEC;

  // alloc device memory
  hipMalloc((void**) &da, size * sizeof(int));
  hipMalloc((void**) &db, size * sizeof(int));
  hipMalloc((void**) &dc, size * sizeof(int));

  // copy input vectors to device
  hipMemcpy(da, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db, b, size * sizeof(int), hipMemcpyHostToDevice);

  // start device timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  add_vectors_gpu<<<num_blocks, num_threads>>>(da, db, dc, size);

  // stop device timer
  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  gpu_time = (double) sdkGetTimerValue(&timer) / 1000;
  sdkDeleteTimer(&timer);

  // copy results from device to host
  hipMemcpy(c_gpu, dc, size * sizeof(int), hipMemcpyDeviceToHost);

  // clean up device memory
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  // print results
  // printf("CPU time: %f\n", cpu_time);
  // printf("GPU time: %f\n", gpu_time);
  // printf("Check: %d\n", check(c_cpu, c_gpu, size));

  printf("%d, %d, %f,%f,%d\n", num_blocks, num_threads, cpu_time, gpu_time, check(c_cpu, c_gpu, size));

  // clean up host memory
  free(a);
  free(b);
  free(c_cpu);
  free(c_gpu);

  return 0;
}

__global__ void add_vectors_gpu(int *a, int *b, int *c, unsigned long long int size) {
  unsigned long long int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(index < size) {
    c[index] = a[index] + b[index];
  }
}

void add_vectors_cpu(int *a, int *b, int *c, unsigned long long int size) {
  unsigned long long int i;
  for (i = 0; i < size; i++) {
    c[i] = a[i] + b[i];
  }
}

int *alloc_vector(unsigned long long int size) {
  return (int *) malloc(sizeof(int) * size);
}

int * random_vector(unsigned long long int size) {
  unsigned long long int i;
  int *vector;
  vector = alloc_vector(size);

  for (i = 0; i < size; i++) {
    vector[i] = rand() % 100;
  }

  return vector;
}

int check(int *c1, int *c2, unsigned long long int size) {
  unsigned long long int i;
  for (i = 0; i < size; i++) {
    if (c1[i] != c2[i]) return 0;
  }
  return 1;
}

void print_vector(int *v, unsigned long long int size) {
  unsigned long long int i;
  for (i = 0; i < size; i++) {
    printf("%d ", v[i]);
  }
  printf("\n");
}
